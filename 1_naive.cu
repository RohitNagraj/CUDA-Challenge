#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

// #define N 100'000'000
#define N 100'000'000
#define BLOCK_SIZE 32
#define N_STREAMS 1

__global__ void transform(float *arr, float *output, float *sum)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < N)
    {
        if ((i % 4 == 0) && (i % 32 < 16))
        {
            output[i] = sinf(arr[i]);
        }
        else if ((i % 4 == 1) && (i % 32 < 16))
        {
            {
                output[i] = cosf(arr[i]);
                if (output[i] > 0.5)
                    atomicAdd(sum, output[i]);
            }
        }
        else if ((i % 4 == 2) && (i % 32 < 16))
        {
            output[i] = logf(arr[i]);
        }
        else if ((i % 4 == 3) && (i % 32 < 16))
        {
            output[i] = expf(arr[i]);
        }
        else if ((i % 4 == 0) && (i % 32 >= 16))
        {
            output[i] = sinf(arr[i]) * sinf(arr[i - 16]);
        }
        else if ((i % 4 == 1) && (i % 32 >= 16))
        {
            output[i] = cosf(arr[i]) * cosf(arr[i - 16]);
        }
        else if ((i % 4 == 2) && (i % 32 >= 16))
        {
            output[i] = logf(arr[i]) * logf(arr[i - 16]);
        }
        else if ((i % 4 == 3) && (i % 32 >= 16))
        {
            output[i] = expf(arr[i]) * expf(arr[i - 16]);
        }
    }
}

__global__ void baseline(float *arr, float *output)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    output[i] = arr[i];
}

int main()
{

    srand(12345);
    float *h_arr, *h_output, h_sum = 0.0;
    float *d_arr, *d_output, *d_sum;
    int i;
    float elapsedTimeBaseline, elapsedTime;

    hipEvent_t startBaseline, stopBaseline, start, stop;

    hipEventCreate(&startBaseline);
    hipEventCreate(&start);
    hipEventCreate(&stopBaseline);
    hipEventCreate(&stop);

    hipHostMalloc(&h_arr, N * sizeof(float), hipHostMallocDefault);
    hipHostMalloc(&h_output, N * sizeof(float), hipHostMallocDefault);

    hipMalloc(&d_arr, N * sizeof(float));
    hipMalloc(&d_output, N * sizeof(float));
    hipMalloc(&d_sum, sizeof(float));

    for (i = 0; i < N; i++)
    {
        h_arr[i] = (float)rand() / RAND_MAX * 5.0f;
    }

    dim3 grid(N / (BLOCK_SIZE * N_STREAMS));
    dim3 block(BLOCK_SIZE);

    // Host to Device Memcpy
    hipMemcpy(d_arr, h_arr, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_sum, &h_sum, sizeof(float), hipMemcpyHostToDevice);

    // Warm-up
    baseline<<<grid, block>>>(d_arr, d_output);
    hipDeviceSynchronize();

    hipEventRecord(startBaseline, 0);
    baseline<<<grid, block>>>(d_arr, d_output);
    hipEventRecord(stopBaseline, 0);
    hipEventSynchronize(stopBaseline);

    hipEventRecord(start, 0);
    transform<<<grid, block>>>(d_arr, d_output, d_sum);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    hipMemcpy(h_output, d_output, N * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(&h_sum, d_sum, sizeof(float), hipMemcpyDeviceToHost);

    hipEventElapsedTime(&elapsedTimeBaseline, startBaseline, stopBaseline);
    printf("Baseline execution time: %.3f ms\n", elapsedTimeBaseline);

    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Kernel execution time: %.3f ms\n", elapsedTime);

    // printf("First 32 inputs and transformations:\n");
    // for (int i = 0; i < 32; i++)
    //     printf("Input: %f, Transformation: %f\n", h_arr[i], h_output[i]);
    printf("Sum of Sin: %f\n", h_sum);

    hipHostFree(h_arr);
    hipHostFree(h_output);

    hipFree(d_arr);
    hipFree(d_output);
    hipFree(d_sum);

    hipEventDestroy(startBaseline);
    hipEventDestroy(stopBaseline);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    return 0;
}