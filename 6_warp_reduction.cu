#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define N (1 << 27)
#define BLOCK_SIZE 256
#define BLOCK_SIZE_BASELINE 256
#define DELTA 0.0001

__global__ void transform(float *arr, float *output, double *sum)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    double thread_sum = 0.0;
    __shared__ double warp_sums[BLOCK_SIZE / 32];

    if (i < N)
    {
        float xi = arr[i];
        float ximinus16 = arr[i - 16];
        int mod4 = (i & 3);
        bool first_block = ((i & 31) < 16);

        output[i] =
            (mod4 == 0) ? (first_block ? sinf(xi) : sinf(xi) * sinf(ximinus16)) : (mod4 == 1) ? (first_block ? cosf(xi) : cosf(xi) * cosf(ximinus16))
                                                                              : (mod4 == 2)   ? (first_block ? logf(xi) : logf(xi) * logf(ximinus16))
                                                                                              : (first_block ? expf(xi) : expf(xi) * expf(ximinus16));

        if ((i % 4 == 1) && (output[i] > 0.5))
            thread_sum = (double)output[i - 1];
    }

    // Reduce within warp
    for (int offset = 16; offset > 0; offset /= 2)
        thread_sum += __shfl_down_sync(0xffffffff, thread_sum, offset);

    // First thread of each warp writes to shared memory
    int warp_id = threadIdx.x / 32;
    if (threadIdx.x % 32 == 0)
        warp_sums[warp_id] = thread_sum;

    __syncthreads();

    // First thread sequentially computes the sums for all warps. I tried tree-based and warp-reduction. They didn't make any performance difference.
    if (threadIdx.x == 0)
    {
        double block_sum = 0.0;
        for (int j = 0; j < BLOCK_SIZE / 32; j++)
            block_sum += warp_sums[j];
        atomicAdd(sum, block_sum);
    }
}

__global__ void baseline(float *arr, float *output)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    output[i] = arr[i];
}

int main()
{

    srand(12345);
    float *h_arr, *h_output;
    double h_sum = 0.0;
    float *d_arr, *d_output;
    double *d_sum;
    int i;
    float elapsedTimeBaseline, elapsedTime;

    hipEvent_t startBaseline, stopBaseline, start, stop;

    hipEventCreate(&startBaseline);
    hipEventCreate(&start);
    hipEventCreate(&stopBaseline);
    hipEventCreate(&stop);

    hipHostMalloc(&h_arr, N * sizeof(float), hipHostMallocDefault);
    hipHostMalloc(&h_output, N * sizeof(float), hipHostMallocDefault);

    hipMalloc(&d_arr, N * sizeof(float));
    hipMalloc(&d_output, N * sizeof(float));
    hipMalloc(&d_sum, sizeof(double));

    for (i = 0; i < N; i++)
    {
        h_arr[i] = (float)rand() / RAND_MAX * 5.0f;
    }

    dim3 grid(N / (BLOCK_SIZE));
    dim3 block(BLOCK_SIZE);

    hipMemcpy(d_arr, h_arr, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_sum, &h_sum, sizeof(float), hipMemcpyHostToDevice);

    // Warm-up
    baseline<<<N / BLOCK_SIZE_BASELINE, BLOCK_SIZE_BASELINE>>>(d_arr, d_output);
    hipDeviceSynchronize();

    hipEventRecord(startBaseline, 0);
    baseline<<<N / BLOCK_SIZE_BASELINE, BLOCK_SIZE_BASELINE>>>(d_arr, d_output);
    hipEventRecord(stopBaseline, 0);
    hipEventSynchronize(stopBaseline);

    hipEventRecord(start, 0);
    transform<<<grid, block>>>(d_arr, d_output, d_sum);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    hipMemcpy(h_output, d_output, N * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(&h_sum, d_sum, sizeof(double), hipMemcpyDeviceToHost);

    // Correctness Testing
    bool correct = true;

    for (int i = 0; i < 64; i++)
    {
        if ((i % 4 == 0) && (i % 32 < 16))
            if (fabsf(h_output[i] - sinf(h_arr[i])) > DELTA)
            {
                correct = false;
                printf("Expected: %f, Got: %f at index: %d\n", sinf(h_arr[i]), h_output[i], i);
            }
        else if ((i % 4 == 1) && (i % 32 < 16))
            if (fabsf(h_output[i] - cosf(h_arr[i])) > DELTA)
            {
                correct = false;
                printf("Expected: %f, Got: %f at index: %d\n", cosf(h_arr[i]), h_output[i], i);
            }
        else if ((i % 4 == 2) && (i % 32 < 16))
            if (fabsf(h_output[i] - logf(h_arr[i])) > DELTA)
            {
                correct = false;
                printf("Expected: %f, Got: %f at index: %d\n", logf(h_arr[i]), h_output[i], i);
            }
        else if ((i % 4 == 3) && (i % 32 < 16))
            if (fabsf(h_output[i] - expf(h_arr[i])) > DELTA)
            {
                correct = false;
                printf("Expected: %f, Got: %f at index: %d\n", expf(h_arr[i]), h_output[i], i);
            }
        else if ((i % 4 == 0) && (i % 32 >= 16))
            if (fabsf(h_output[i] - (sinf(h_arr[i]) * sinf(h_arr[i - 16]))) > DELTA)
            {
                correct = false;
                printf("Expected: %f, Got: %f at index: %d\n", (sinf(h_arr[i]) * sinf(h_arr[i - 16])), h_output[i], i);
            }
        else if ((i % 4 == 1) && (i % 32 >= 16))
            if (fabsf(h_output[i] - (cosf(h_arr[i]) * cosf(h_arr[i - 16]))) > DELTA)
            {
                correct = false;
                printf("Expected: %f, Got: %f at index: %d\n", (cosf(h_arr[i]) * cosf(h_arr[i - 16])), h_output[i], i);
            }
        else if ((i % 4 == 2) && (i % 32 >= 16))
            if (fabsf(h_output[i] - (logf(h_arr[i]) * logf(h_arr[i - 16]))) > DELTA)
            {
                correct = false;
                printf("Expected: %f, Got: %f at index: %d\n", (logf(h_arr[i]) * logf(h_arr[i - 16])), h_output[i], i);
            }
        else if ((i % 4 == 3) && (i % 32 >= 16))
            if (fabsf(h_output[i] - (expf(h_arr[i]) * expf(h_arr[i - 16]))) > DELTA)
            {
                correct = false;
                printf("Expected: %f, Got: %f at index: %d\n", (expf(h_arr[i]) * expf(h_arr[i - 16])), h_output[i], i);
            }
    }
    if (!correct)
        printf("CORRECTNESS TEST FAILED!\n");
    else
        printf("Correctness Tests Passed!\n");
    printf("Sum - Actual: %f, Expected: 566300.125\n", (float)h_sum);

    hipEventElapsedTime(&elapsedTimeBaseline, startBaseline, stopBaseline);
    printf("Baseline execution time: %.3f ms\n", elapsedTimeBaseline);

    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Kernel execution time: %.3f ms\n", elapsedTime);

    printf("Speed: %f%%\n", (elapsedTimeBaseline / elapsedTime) * 100);

    double bytes_baseline = 2.0 * N * sizeof(float);
    double bandwidth_baseline = (bytes_baseline / 1e9) / (elapsedTimeBaseline / 1000.0);
    printf("Baseline Bandwidth: %.2f GB/s\n", bandwidth_baseline);

    double bytes_transform = 2.5 * N * sizeof(float);
    double bandwidth_transform = (bytes_transform / 1e9) / (elapsedTime / 1000.0);
    printf("Kernel Bandwidth: %.2f GB/s\n", bandwidth_transform);

    hipHostFree(h_arr);
    hipHostFree(h_output);

    hipFree(d_arr);
    hipFree(d_output);
    hipFree(d_sum);

    hipEventDestroy(startBaseline);
    hipEventDestroy(stopBaseline);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    return 0;
}