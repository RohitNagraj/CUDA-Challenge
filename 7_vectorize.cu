#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define N (1 << 27)
#define BLOCK_SIZE 256
#define BLOCK_SIZE_BASELINE 256
#define DELTA 0.0001

__global__ void transform(float *arr, float *output, double *sum)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int i = tid * 4; // Each thread processes 4 elements

    double thread_sum = 0.0;
    __shared__ double warp_sums[BLOCK_SIZE / 32];

    if (i + 3 < N)
    {
        // Load 4 floats at once using float4
        float4 xi_vec = *reinterpret_cast<float4 *>(&arr[i]);
        float4 output_vec;

        for (int j = 0; j < 4; j++)
        {
            int idx = i + j;
            float xi = ((float *)&xi_vec)[j];
            int mod4 = (idx & 3);
            bool first_block = ((idx & 31) < 16);

            float result;
            if (first_block)
            {
                if (mod4 == 0)
                    result = sinf(xi);
                else if (mod4 == 1)
                    result = cosf(xi);
                else if (mod4 == 2)
                    result = logf(xi);
                else
                    result = expf(xi);
            }
            else
            {
                float ximinus16 = arr[idx - 16];
                if (mod4 == 0)
                    result = sinf(xi) * sinf(ximinus16);
                else if (mod4 == 1)
                    result = cosf(xi) * cosf(ximinus16);
                else if (mod4 == 2)
                    result = logf(xi) * logf(ximinus16);
                else
                    result = expf(xi) * expf(ximinus16);
            }

            ((float *)&output_vec)[j] = result;

            // Accumulate for sum (when idx % 4 == 1 and result > 0.5)
            if ((idx % 4 == 1) && (result > 0.5))
            {
                // Need to get output[idx - 1]
                if (j > 0)
                    thread_sum += (double)((float *)&output_vec)[j - 1];
                else if (idx > 0)
                    thread_sum += (double)output[idx - 1];
            }
        }

        // Store 4 floats at once using float4
        *reinterpret_cast<float4 *>(&output[i]) = output_vec;
    }

    // Reduce within warp
    for (int offset = 16; offset > 0; offset /= 2)
        thread_sum += __shfl_down_sync(0xffffffff, thread_sum, offset);

    // First thread of each warp writes to shared memory
    int warp_id = threadIdx.x / 32;
    if (threadIdx.x % 32 == 0)
        warp_sums[warp_id] = thread_sum;

    __syncthreads();

    // First warp reduces the warp sums
    if (threadIdx.x == 0)
    {
        double block_sum = 0.0;
        for (int j = 0; j < BLOCK_SIZE / 32; j++)
            block_sum += warp_sums[j];
        atomicAdd(sum, block_sum);
    }
}

__global__ void baseline(float *arr, float *output)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    output[i] = arr[i];
}

int main()
{
    srand(12345);
    float *h_arr, *h_output;
    double h_sum = 0.0;
    float *d_arr, *d_output;
    double *d_sum;
    int i;
    float elapsedTimeBaseline, elapsedTime;

    hipEvent_t startBaseline, stopBaseline, start, stop;

    hipEventCreate(&startBaseline);
    hipEventCreate(&start);
    hipEventCreate(&stopBaseline);
    hipEventCreate(&stop);

    hipHostMalloc(&h_arr, N * sizeof(float), hipHostMallocDefault);
    hipHostMalloc(&h_output, N * sizeof(float), hipHostMallocDefault);

    hipMalloc(&d_arr, N * sizeof(float));
    hipMalloc(&d_output, N * sizeof(float));
    hipMalloc(&d_sum, sizeof(double));

    for (i = 0; i < N; i++)
    {
        h_arr[i] = (float)rand() / RAND_MAX * 5.0f;
    }

    // Each thread processes 4 elements, so divide grid size by 4
    dim3 grid(N / (BLOCK_SIZE * 4));
    dim3 block(BLOCK_SIZE);

    hipMemcpy(d_arr, h_arr, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_sum, &h_sum, sizeof(double), hipMemcpyHostToDevice);

    // Warm-up
    baseline<<<N / BLOCK_SIZE_BASELINE, BLOCK_SIZE_BASELINE>>>(d_arr, d_output);
    hipDeviceSynchronize();

    hipEventRecord(startBaseline, 0);
    baseline<<<N / BLOCK_SIZE_BASELINE, BLOCK_SIZE_BASELINE>>>(d_arr, d_output);
    hipEventRecord(stopBaseline, 0);
    hipEventSynchronize(stopBaseline);

    hipEventRecord(start, 0);
    transform<<<grid, block>>>(d_arr, d_output, d_sum);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    hipMemcpy(h_output, d_output, N * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(&h_sum, d_sum, sizeof(double), hipMemcpyDeviceToHost);

    // Correctness Testing
    bool correct = true;

    for (int i = 0; i < 64; i++)
    {
        if ((i % 4 == 0) && (i % 32 < 16))
            if (fabsf(h_output[i] - sinf(h_arr[i])) > DELTA)
            {
                correct = false;
                printf("Expected: %f, Got: %f at index: %d\n", sinf(h_arr[i]), h_output[i], i);
            }
        else if ((i % 4 == 1) && (i % 32 < 16))
            if (fabsf(h_output[i] - cosf(h_arr[i])) > DELTA)
            {
                correct = false;
                printf("Expected: %f, Got: %f at index: %d\n", cosf(h_arr[i]), h_output[i], i);
            }
        else if ((i % 4 == 2) && (i % 32 < 16))
            if (fabsf(h_output[i] - logf(h_arr[i])) > DELTA)
            {
                correct = false;
                printf("Expected: %f, Got: %f at index: %d\n", logf(h_arr[i]), h_output[i], i);
            }
        else if ((i % 4 == 3) && (i % 32 < 16))
            if (fabsf(h_output[i] - expf(h_arr[i])) > DELTA)
            {
                correct = false;
                printf("Expected: %f, Got: %f at index: %d\n", expf(h_arr[i]), h_output[i], i);
            }
        else if ((i % 4 == 0) && (i % 32 >= 16))
            if (fabsf(h_output[i] - (sinf(h_arr[i]) * sinf(h_arr[i - 16]))) > DELTA)
            {
                correct = false;
                printf("Expected: %f, Got: %f at index: %d\n", (sinf(h_arr[i]) * sinf(h_arr[i - 16])), h_output[i], i);
            }
        else if ((i % 4 == 1) && (i % 32 >= 16))
            if (fabsf(h_output[i] - (cosf(h_arr[i]) * cosf(h_arr[i - 16]))) > DELTA)
            {
                correct = false;
                printf("Expected: %f, Got: %f at index: %d\n", (cosf(h_arr[i]) * cosf(h_arr[i - 16])), h_output[i], i);
            }
        else if ((i % 4 == 2) && (i % 32 >= 16))
            if (fabsf(h_output[i] - (logf(h_arr[i]) * logf(h_arr[i - 16]))) > DELTA)
            {
                correct = false;
                printf("Expected: %f, Got: %f at index: %d\n", (logf(h_arr[i]) * logf(h_arr[i - 16])), h_output[i], i);
            }
        else if ((i % 4 == 3) && (i % 32 >= 16))
            if (fabsf(h_output[i] - (expf(h_arr[i]) * expf(h_arr[i - 16]))) > DELTA)
            {
                correct = false;
                printf("Expected: %f, Got: %f at index: %d\n", (expf(h_arr[i]) * expf(h_arr[i - 16])), h_output[i], i);
            }
    }
    if (!correct)
        printf("CORRECTNESS TEST FAILED!\n");
    else
        printf("Correctness Tests Passed!\n");
    printf("Sum - Actual: %f, Expected: 566300.125\n", (float)h_sum);

    hipEventElapsedTime(&elapsedTimeBaseline, startBaseline, stopBaseline);
    printf("Baseline execution time: %.3f ms\n", elapsedTimeBaseline);

    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Kernel execution time: %.3f ms\n", elapsedTime);

    printf("Speed: %f%%\n", (elapsedTimeBaseline / elapsedTime) * 100);

    double bytes_baseline = 2.0 * N * sizeof(float);
    double bandwidth_baseline = (bytes_baseline / 1e9) / (elapsedTimeBaseline / 1000.0);
    printf("Baseline Bandwidth: %.2f GB/s\n", bandwidth_baseline);

    double bytes_transform = 2.5 * N * sizeof(float);
    double bandwidth_transform = (bytes_transform / 1e9) / (elapsedTime / 1000.0);
    printf("Kernel Bandwidth: %.2f GB/s\n", bandwidth_transform);

    hipHostFree(h_arr);
    hipHostFree(h_output);

    hipFree(d_arr);
    hipFree(d_output);
    hipFree(d_sum);

    hipEventDestroy(startBaseline);
    hipEventDestroy(stopBaseline);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    return 0;
}